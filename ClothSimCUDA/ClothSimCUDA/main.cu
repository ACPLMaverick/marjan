#include "Common.h"
#include "System.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <conio.h>

int main()
{
	int error;
	
	error = System::GetInstance()->Initialize();

	if (error != CS_ERR_NONE)
	{
#ifdef _DEBUG
		printf("\nSystem.Initialize exited with error code %d\n", error);
		getch();
#endif
		return error;
	}

	error = System::GetInstance()->Run();

	if (error != CS_ERR_NONE)
	{
#ifdef _DEBUG
		printf("\nSystem.Run exited with error code %d\n", error);
		getch();
#endif
		return error;
	}

	error = System::GetInstance()->Shutdown();

	if (error != CS_ERR_NONE)
	{
#ifdef _DEBUG
		printf("\nSystem.Shutdown exited with error code %d\n", error);
		getch();
#endif
		return error;
	}

	System::DestroyInstance();

	printf("\nProgram terminated successfully.\n");
	getch();
	return CS_ERR_NONE;
}

