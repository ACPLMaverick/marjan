#include "hip/hip_runtime.h"

#include "clothSpringSimulation.h"


__global__ void CalculateSpringsKernel(
	Vertex* vertPtr, 
	Spring* springPtr, 
	glm::vec3* posPtr, 
	glm::vec3* nrmPtr, 
	glm::vec4* colPtr, 
	const float* grav,
	const int N
	)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int v_cur = (i * gridDim.y * blockDim.y) + j;

	if (v_cur >= N)
		return;

	posPtr[springPtr[v_cur].idFirst].y = posPtr[springPtr[v_cur].idFirst].y - 0.00055f;
	//posPtr[springPtr[v_cur].idSecond].y -= 0.00055f;
}

__global__ void CalculateForcesKernel(
	Vertex* vertPtr, 
	Spring* springPtr, 
	glm::vec3* posPtr, 
	glm::vec3* nrmPtr, 
	glm::vec4* colPtr, 
	const float* grav, 
	const float delta,
	const unsigned int N
	)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int v_cur = (i * gridDim.y * blockDim.y) + j;

	if (v_cur >= N)
		return;
	
	vertPtr[v_cur].force = glm::vec3(0.0f, 0.0f, 0.0f);
	vertPtr[v_cur].velocity = glm::vec3(0.0f, 0.0f, 0.0f);
	float el = 0.1f;
	int id = vertPtr[v_cur].id;

	// calculate elasticity force for each neighbouring vertices
	for (int i = 0; i < VERTEX_NEIGHBOURING_VERTICES; ++i)
	{
		vertPtr[v_cur].force.x +=
			-el * (abs(posPtr[id].x - posPtr[vertPtr[v_cur].neighbours[i]].x) - vertPtr[v_cur].springLengths[i]) *
			(posPtr[id].x - posPtr[vertPtr[v_cur].neighbours[i]].x) /
			((abs(posPtr[id].x - posPtr[vertPtr[v_cur].neighbours[i]].x))) * 
			vertPtr[v_cur].neighbourMultipliers[i];

		vertPtr[v_cur].force.y +=
			-el * (abs(posPtr[id].y - posPtr[vertPtr[v_cur].neighbours[i]].y) - vertPtr[v_cur].springLengths[i]) *
			(posPtr[id].y - posPtr[vertPtr[v_cur].neighbours[i]].y) /
			((abs(posPtr[id].y - posPtr[vertPtr[v_cur].neighbours[i]].y))) *
			vertPtr[v_cur].neighbourMultipliers[i];

		vertPtr[v_cur].force.z +=
			-el * (abs(posPtr[id].z - posPtr[vertPtr[v_cur].neighbours[i]].z) - vertPtr[v_cur].springLengths[i]) *
			(posPtr[id].z - posPtr[vertPtr[v_cur].neighbours[i]].z) /
			((abs(posPtr[id].z - posPtr[vertPtr[v_cur].neighbours[i]].z))) *
			vertPtr[v_cur].neighbourMultipliers[i];
	}


	// calculate gravity force
	vertPtr[v_cur].force +=
		vertPtr[v_cur].mass * glm::vec3(0.0f, -(*grav) / 10.0f, 0.0f);

	// calculate air damp force
	vertPtr[v_cur].force +=
		-vertPtr[v_cur].dampCoeff * vertPtr[v_cur].velocity;

	// ?calculate repulsive force?

	// check hooks
	vertPtr[v_cur].force *= vertPtr[v_cur].lockMultiplier;


	// calculate acceleration and use Verelet integration to calculate position
	glm::vec3 newPos;
	glm::vec3 acc = vertPtr[v_cur].force / vertPtr[v_cur].mass;

	newPos = 2.0f * posPtr[id] - vertPtr[v_cur].prevPosition + acc * delta * delta;
	vertPtr[v_cur].prevPosition = posPtr[id];
	posPtr[id] = newPos;

	// update velocity
	vertPtr[v_cur].velocity = (newPos - vertPtr[v_cur].prevPosition) / delta;
}

__global__ void CalculatePositionsKernel(
	Vertex* vertPtr, 
	Spring* springPtr, 
	glm::vec3* posPtr, 
	glm::vec3* nrmPtr, 
	glm::vec4* colPtr, 
	const float* grav,
	const int N
	)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int v_cur = (i * gridDim.y * blockDim.y) + j;

	if (v_cur >= N)
		return;
}

__global__ void CalculateNormalsKernel(
	Vertex* vertPtr, 
	Spring* springPtr, 
	glm::vec3* posPtr, 
	glm::vec3* nrmPtr, 
	glm::vec4* colPtr, 
	const float* grav, 
	const int N
	)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int v_cur = (i * gridDim.y * blockDim.y) + j;

	if (v_cur >= N)
		return;
}

//////////////////////////////////////////////////////

clothSpringSimulation::clothSpringSimulation()
{
}

clothSpringSimulation::~clothSpringSimulation()
{

}

unsigned int clothSpringSimulation::ClothSpringSimulationInitialize(
	unsigned int vertexPositionSize,
	unsigned int vertexNormalSize,
	unsigned int vertexColorSize,
	unsigned int edgesWidth,
	unsigned int edgesLength,
	glm::vec3* vertexPositionPtr,
	glm::vec3* vertexNormalPtr,
	glm::vec4* vertexColorPtr
	)
{
	hipError_t cudaStatus;

	// save data given to function
	m_vertexPositionSize = vertexPositionSize;
	m_vertexNormalSize = vertexNormalSize;
	m_vertexColorSize = vertexColorSize;
	m_allEdgesWidth = edgesWidth;
	m_allEdgesLength = edgesLength;
	m_vertexCount = m_allEdgesLength * m_allEdgesWidth;
	m_springCount = (m_allEdgesLength - 1) * (m_allEdgesWidth) + (m_allEdgesWidth - 1) * m_allEdgesLength;
	m_posPtr = vertexPositionPtr;
	m_nrmPtr = vertexNormalPtr;
	m_colPtr = vertexColorPtr;

	// generate vertex and spring arrays, to help with computations

	m_vertices = new Vertex[m_vertexCount];

	for (int i = 0; i < m_vertexCount; ++i)
	{
		m_vertices[i].id = i;
		m_vertices[i].mass = VERTEX_MASS;
		m_vertices[i].lockMultiplier = 1.0f;
		m_vertices[i].prevPosition = m_posPtr[i];
		m_vertices[i].dampCoeff = VERTEX_DAMP;

		// calculating neighbouring vertices ids and spring lengths

		float lengthX = abs(m_posPtr[0].x - m_posPtr[m_vertexCount - 1].x) / (float)(m_allEdgesLength - 1);
		float lengthZ = abs(m_posPtr[0].z - m_posPtr[m_vertexCount - 1].z) / (float)(m_allEdgesWidth - 1);

		// upper
		m_vertices[i].neighbours[0] = (i - 1) % m_vertexCount;
		if (i % m_allEdgesLength)
		{
			m_vertices[i].neighbourMultipliers[0] = 1.0f;
			m_vertices[i].springLengths[0] = lengthZ;
		}	
		else
		{
			m_vertices[i].neighbourMultipliers[0] = 0.0f;
			m_vertices[i].springLengths[0] = 0.0f;
		}

		// lower
		m_vertices[i].neighbours[1] = (i + 1) % m_vertexCount;
		if (i % m_allEdgesLength != (m_allEdgesLength - 1))
		{
			m_vertices[i].neighbourMultipliers[1] = 1.0f;
			m_vertices[i].springLengths[1] = lengthZ;
		}
		else
		{
			m_vertices[i].neighbourMultipliers[1] = 0.0f;
			m_vertices[i].springLengths[1] = 0.0f;
		}

		// left
		m_vertices[i].neighbours[2] = (i - m_allEdgesLength) % m_vertexCount;
		if (i >= m_allEdgesLength)
		{
			m_vertices[i].neighbourMultipliers[2] = 1.0f;
			m_vertices[i].springLengths[1] = lengthX;
		}
		else
		{
			m_vertices[i].neighbourMultipliers[2] = 0.0f;
			m_vertices[i].springLengths[2] = 0.0f;
		}

		// right
		m_vertices[i].neighbours[3] = (i + m_allEdgesLength) % m_vertexCount;
		if (i < (m_vertexCount - m_allEdgesLength))
		{
			m_vertices[i].neighbourMultipliers[3] = 1.0f;
			m_vertices[i].springLengths[3] = lengthX;
		}
		else
		{
			m_vertices[i].neighbourMultipliers[3] = 0.0f;
			m_vertices[i].springLengths[3] = 0.0f;
		}
	}


	m_springs = new Spring[m_springCount];

	for (int i = 0, s = 0; i < m_vertexCount; ++i)
	{
		// do I create "lower" spring?
		if (i % m_allEdgesLength != (m_allEdgesLength - 1))
		{
			m_springs[s].idFirst = i;
			m_springs[s].idSecond = i + 1;

			m_springs[s].baseLength = glm::length(m_posPtr[i] - m_posPtr[i + 1]);
			m_springs[s].elasticity = SPRING_ELASTICITY;

			++s;
		}

		// do I create "right" spring?
		if (i < (m_vertexCount - m_allEdgesLength))
		{
			m_springs[s].idFirst = i;
			m_springs[s].idSecond = i + m_allEdgesLength;

			m_springs[s].baseLength = glm::length(m_posPtr[i] - m_posPtr[i + 1]);
			m_springs[s].elasticity = SPRING_ELASTICITY;

			++s;
		}
	}

	//////////////////////////////

	// Get Device info
	m_deviceProperties = new hipDeviceProp_t;
	cudaStatus = hipGetDeviceProperties(m_deviceProperties, 0);
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipGetDeviceProperties failed!  Do you have a CUDA-capable GPU installed?");
		FreeMemory();
		return cudaStatus;
	}

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		FreeMemory();
		return cudaStatus;
	}

	// Allocate GPU buffers for six vectors (2 input for helpers, 3 input for buffers) and one float (gravity)
	cudaStatus = hipMalloc((void**)&i_vertexPtr, m_vertexCount * sizeof(Vertex));
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipMalloc for vertex helper buffer failed!");
		FreeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&i_springPtr, m_springCount * sizeof(Spring));
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipMalloc for spring helper buffer failed!");
		FreeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&i_posPtr, m_vertexCount * m_vertexPositionSize);
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipMalloc for position buffer failed!");
		FreeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&i_nrmPtr, m_vertexCount * m_vertexNormalSize);
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipMalloc for normal buffer failed!");
		FreeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&i_colPtr, m_vertexCount * m_vertexColorSize);
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipMalloc for color buffer failed!");
		FreeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&i_gravPtr, sizeof(float));
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipMalloc for gravity variable failed!");
		FreeMemory();
		return cudaStatus;
	}

	// copy helper buffers to device memory
	cudaStatus = hipMemcpy(i_vertexPtr, m_vertices, m_vertexCount * sizeof(Vertex), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(i_springPtr, m_springs, m_springCount * sizeof(Spring), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return cudaStatus;
	}
}

unsigned int clothSpringSimulation::ClothSpringSimulationUpdate(float gravity, double delta)
{
	// Add vectors in parallel.
	hipError_t cudaStatus = CalculateForces(gravity, delta);
	if (cudaStatus != hipSuccess) {
		printf("CUDA: AddWithCuda failed!");
		return CS_ERR_CLOTHSIMULATOR_CUDA_FAILED;
	}

	//printf("%f\n",m_posPtr[0].y);
	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
	//	c[0], c[1], c[2], c[3], c[4]);

	return CS_ERR_NONE;
}

unsigned int clothSpringSimulation::ClothSpringSimulationShutdown()
{
	hipError_t cudaStatus;

	delete m_deviceProperties;

	FreeMemory();

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipDeviceReset failed!");
		return 1;
	}
}


/////////////////////////////////////////////////////


/////////////////////////////////////////////////////


inline hipError_t clothSpringSimulation::CalculateForces(float gravity, double delta)
{
	hipError_t status;

	// copy vertex data and simulation variables to device memory

	status = hipMemcpy(i_vertexPtr, m_vertices, m_vertexCount * sizeof(Vertex), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return status;
	}

	/*
	status = hipMemcpy(i_springPtr, m_springs, m_springCount * sizeof(Spring), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return status;
	}
	*/

	status = hipMemcpy(i_posPtr, m_posPtr, m_vertexCount * m_vertexPositionSize, hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return status;
	}

	status = hipMemcpy(i_nrmPtr, m_nrmPtr, m_vertexCount * m_vertexNormalSize, hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return status;
	}

	status = hipMemcpy(i_colPtr, m_colPtr, m_vertexCount * m_vertexColorSize, hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return status;
	}

	status = hipMemcpy(i_gravPtr, &gravity, sizeof(float), hipMemcpyHostToDevice);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return status;
	}

	// launch kernel
	int p = m_deviceProperties->warpSize;
	int sX = (m_allEdgesWidth - 1) * m_allEdgesLength;
	int sY = (m_allEdgesLength - 1) * m_allEdgesWidth;
	dim3 gridVerts((m_allEdgesWidth + p - 1) / p, (m_allEdgesLength + p - 1) / p, 1);
	dim3 gridSprings((sX + p - 1) / p, (sY + p - 1) / p, 1);
	dim3 blockVerts(p, p, 1);
	dim3 blockSprings(p, p, 1);

	//CalculateSpringsKernel << < gridSprings, blockSprings >> > (i_vertexPtr, i_springPtr, i_posPtr, i_nrmPtr, i_colPtr, i_gravPtr, m_springCount);
	CalculateForcesKernel << < gridVerts, blockVerts >> > (i_vertexPtr, i_springPtr, i_posPtr, i_nrmPtr, i_colPtr, i_gravPtr, delta / 1000.0f, m_vertexCount);
	//CalculatePositionsKernel << < gridVerts, blockVerts >> > (i_vertexPtr, i_springPtr, i_posPtr, i_nrmPtr, i_colPtr, i_gravPtr, m_vertexCount);
	//CalculateNormalsKernel << < gridVerts, blockVerts >> > (i_vertexPtr, i_springPtr, i_posPtr, i_nrmPtr, i_colPtr, i_gravPtr, m_vertexCount);

	// Check for any errors launching the kernel
	status = hipGetLastError();
	if (status != hipSuccess) {
		fprintf(stderr, "AddKernel launch failed: %s\n", hipGetErrorString(status));
		FreeMemory();
		return status;
	}

	// copy calculated data out of device memory


	status = hipMemcpy(m_vertices, i_vertexPtr, m_vertexCount * sizeof(Vertex), hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return status;
	}

	/*
	status = hipMemcpy(m_springs, i_springPtr, m_springCount * sizeof(Spring), hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return status;
	}
	*/

	status = hipMemcpy(m_posPtr, i_posPtr, m_vertexCount * m_vertexPositionSize, hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return status;
	}

	status = hipMemcpy(m_nrmPtr, i_nrmPtr, m_vertexCount * m_vertexNormalSize, hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return status;
	}

	status = hipMemcpy(m_colPtr, i_colPtr, m_vertexCount * m_vertexColorSize, hipMemcpyDeviceToHost);
	if (status != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return status;
	}

	return status;
}


void clothSpringSimulation::FreeMemory()
{
	hipFree(i_vertexPtr);
	hipFree(i_springPtr);
	hipFree(i_posPtr);
	hipFree(i_nrmPtr);
	hipFree(i_colPtr);
	hipFree(i_gravPtr);
}