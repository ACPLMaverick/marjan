#include "hip/hip_runtime.h"

#include "clothSpringSimulation.h"


__global__ void AddKernel(int *c, const int *a, const int *b)
{
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

//////////////////////////////////////////////////////

clothSpringSimulation::clothSpringSimulation()
{
}

clothSpringSimulation::~clothSpringSimulation()
{

}

unsigned int clothSpringSimulation::ClothSpringSimulationInitialize(unsigned int vertexCount, unsigned int vertexPositionSize, unsigned int vertexNormalSize, unsigned int vertexColorSize)
{
	hipError_t cudaStatus;

	// save data given to function
	m_vertexCount = vertexCount;
	m_vertexPositionSize = vertexPositionSize;
	m_vertexNormalSize = vertexNormalSize;
	m_vertexColorSize = vertexColorSize;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		FreeMemory();
		return cudaStatus;
	}

	// Allocate GPU buffers for six vectors (3 input, 3 output) and one float (gravity)
	cudaStatus = hipMalloc((void**)&dev_c, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipMalloc failed!");
		FreeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_a, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipMalloc failed!");
		FreeMemory();
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_b, arraySize * sizeof(int));
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipMalloc failed!");
		FreeMemory();
		return cudaStatus;
	}
}

unsigned int clothSpringSimulation::ClothSpringSimulationUpdate(glm::vec3* vertexPositionPtr, glm::vec3* vertexNormalPtr, glm::vec3* vertexColorPtr, float gravity)
{
	// Add vectors in parallel.
	hipError_t cudaStatus = AddWithCuda(c, a, b, arraySize);
	if (cudaStatus != hipSuccess) {
		printf("CUDA: AddWithCuda failed!");
		return CS_ERR_CLOTHSIMULATOR_CUDA_FAILED;
	}

	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
	//	c[0], c[1], c[2], c[3], c[4]);

	return CS_ERR_NONE;
}

unsigned int clothSpringSimulation::ClothSpringSimulationShutdown()
{
	hipError_t cudaStatus;

	FreeMemory();

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		printf("CUDA: hipDeviceReset failed!");
		return 1;
	}
}


/////////////////////////////////////////////////////


/////////////////////////////////////////////////////


// Helper function for using CUDA to add vectors in parallel.
inline hipError_t clothSpringSimulation::AddWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
	hipError_t cudaStatus;

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return cudaStatus;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return cudaStatus;
    }

    // Launch a kernel on the GPU with one thread for each element.
    AddKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "AddKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		FreeMemory();
		return cudaStatus;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching AddKernel!\n", cudaStatus);
		FreeMemory();
		return cudaStatus;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
		FreeMemory();
		return cudaStatus;
    }
    
	//FreeMemory();
	return cudaStatus;
}

void clothSpringSimulation::FreeMemory()
{
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);
}