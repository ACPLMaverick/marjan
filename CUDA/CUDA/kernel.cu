#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <conio.h>

#include "Bitmap.h"

#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>

hipError_t BoxBlur(uchar3* dataPtr, unsigned int width, unsigned int height, unsigned int level);

__global__ void BlurKernel(uchar3* inputData, uchar3* outputData, unsigned int width, unsigned int height, unsigned int level)
{
    unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;

	int x = offset % width;
	int y = (offset - x) / width;
	unsigned int size = width*height;
	int iLevel = (int)level;

		float oR = 0.0f, oG = 0.0f, oB = 0.0f;
		unsigned int sum = 0;

		for (int i = -iLevel; i < iLevel + 1; ++i)
		{
			for (int j = -iLevel; j < iLevel + 1; ++j)
			{
				if ((x + i) >= 0 &&
					(x + i) < width &&
					(y + j) >= 0 &&
					(y + j) < height)
				{
					const int currentOffset = (offset + i + j * width);
					oR += ((float)inputData[currentOffset].z / 255.0f);
					oG += ((float)inputData[currentOffset].y / 255.0f);
					oB += ((float)inputData[currentOffset].x / 255.0f);
					++sum;
				}
				
			}
		}
		outputData[offset].z = (unsigned char)(oR / sum * 255.0f);
		outputData[offset].y = (unsigned char)(oG / sum * 255.0f);
		outputData[offset].x = (unsigned char)(oB / sum * 255.0f);
}

int main()
{
	unsigned int blurLevel;
	char path[64];
	char level[4];
	printf("HIGH PERFORMANCE CUDA-POWERED ULTRA BOX BLUR. \nGive file path: ");
	gets(path);
	printf("Give blur amount: ");
	gets(level);
	blurLevel = strtoul(level, nullptr, 0);

	Bitmap bitmap;
	bool bitmapResult;
	bitmapResult = bitmap.Load("canteen.bmp");
	if (!bitmapResult)
	{
		fprintf(stderr, "Bitmap loading failed!");
		return 1;
	}


    // Add vectors in parallel.
	hipError_t cudaStatus = BoxBlur(bitmap.GetPtr(), bitmap.GetWidth(), bitmap.GetHeight(), blurLevel);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "BoxBlur failed!");
		getch();
        return 1;
    }

	printf("BoxBlur succeeded!");

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
		getch();
        return 1;
    }


	bitmapResult = bitmap.Save("canteen_b.bmp");
	if (!bitmapResult)
	{
		fprintf(stderr, "Bitmap saving failed!");
		getch();
		return 1;
	}

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t BoxBlur(uchar3* dataPtr, unsigned int width, unsigned int height, unsigned int level)
{
	uchar3* input;
	uchar3* output;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&input, width * height * sizeof(uchar3));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(input, dataPtr, width * height* sizeof(uchar3), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&output, width * height* sizeof(uchar3));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	dim3 blockDims(1024, 1, 1);
	dim3 gridDims((unsigned int)ceil((double)(width * height / blockDims.x)), 1, 1);
	//unsigned int nThreads = 64;
    // Launch a kernel on the GPU with one thread for each element.
    BlurKernel KERNEL_ARGS2(blockDims, gridDims) (input, output, width, height, level);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(dataPtr, output, width * height* sizeof(uchar3), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	hipFree(input);
	hipFree(output);
    
    return cudaStatus;
}
